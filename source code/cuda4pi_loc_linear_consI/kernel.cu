#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "definitions.h"
#include "kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>


__global__ void kernel_calcPSFPixel(float *SampledPSF, int SizeX, int SizeY, float *PSFs, float *dPSFx, float *dPSFy, float *dPSFz, float * X, float *Y, float *Z,
	float SampleSpacingXY, float SampleSpacingZ, float StartX, float StartY, float StartZ, int N_int, int PSFSizeOut, int NPSFs)

{
	__shared__ float PSF_Row_Samples[64], dPSFx_Row_Samples[64], dPSFy_Row_Samples[64], dPSFz_Row_Samples[64];	// store the interpolated pixels in here
	__shared__ float PSF_Row_Sum[16], dPSFx_Row_Sum[16], dPSFy_Row_Sum[16], dPSFz_Row_Sum[16];		// This is the sum across rows and columns of interpolated pixels
	 
	__shared__ float theta[5];				// X, Y, Z, I, Bg

	int idX = threadIdx.x;
	int PixelNumber = threadIdx.y;
	float F2d[2], X1z[2], dF2d_y[2], dF2d_x[2];
	float a, b;	// coefficient for linear interpolation
	//Get XYZ
	if ((threadIdx.x == 0) && (threadIdx.y == 0)){
		theta[0] = X[blockIdx.y] * SampleSpacingXY * N_int;
		theta[1] = Y[blockIdx.y] * SampleSpacingXY * N_int;
		theta[2] = Z[blockIdx.y];
	}

	__syncthreads();

	float Y_thread = (N_int*PixelNumber + idX + 0.5)*SampleSpacingXY - theta[1];
	
	int YBaseIndex = floor((Y_thread - StartY) / SampleSpacingXY);

	float SampleSpacingXYInv = 1 / SampleSpacingXY;
	float SampleSpacingZInv = 1 / SampleSpacingZ;
	int idZ = round((theta[2] - StartZ) * SampleSpacingZInv);


	//intialize PSF_Row_Sum counter
	if (threadIdx.x == 0)
	{
		PSF_Row_Sum[threadIdx.y] = 0;
		dPSFx_Row_Sum[threadIdx.y] = 0;
		dPSFy_Row_Sum[threadIdx.y] = 0;
		dPSFz_Row_Sum[threadIdx.y] = 0;
	}
	for (int ii = 0; ii < N_int; ii++) //go right in row
	{
		float X_thread = (blockIdx.x*N_int + ii + 0.5)*SampleSpacingXY - theta[0];


		//for interpolation we need the four surrounding points
		
		int XBaseIndex = floor((X_thread - StartX) * SampleSpacingXYInv);

		//using the follwing notation:
		//X1    X2
		//   o
		//X3	X4

		//These are values of the sampled points.
		for (int nn = 0; nn < 2; nn++)
		{

			int tmp2 = SizeY*SizeX*(idZ+nn) + SizeY*XBaseIndex + YBaseIndex;
			float F1 = SampledPSF[tmp2];
			float F2 = SampledPSF[tmp2 + SizeX];
			float F3 = SampledPSF[tmp2 + 1];
			float F4 = SampledPSF[tmp2 + SizeX + 1];

			//These are locations of the sampled points
			float X1x = XBaseIndex*SampleSpacingXY + StartX;
			float X1y = YBaseIndex*SampleSpacingXY + StartY;
			X1z[nn] = (idZ + nn)*SampleSpacingZ + StartZ;
			//Bilinear interpolation
			gencoeff(F1, F2, SampleSpacingXY, X1x, X1x + SampleSpacingXY,&a,&b);
			float X1X2 = evallinear(a,b,X_thread);
			float dX1X2_x = a;
			gencoeff(F3, F4, SampleSpacingXY, X1x, X1x + SampleSpacingXY, &a, &b);
			float X3X4 = evallinear(a, b, X_thread);
			float dX3X4_x = a;
			gencoeff(X1X2, X3X4, SampleSpacingXY, X1y, X1y + SampleSpacingXY, &a, &b);
			F2d[nn] = evallinear(a, b, Y_thread);
			dF2d_y[nn] = a;
			gencoeff(dX1X2_x, dX3X4_x, SampleSpacingXY, X1y, X1y + SampleSpacingXY, &a, &b);
			dF2d_x[nn] = evallinear(a, b, Y_thread);
		}
		gencoeff(F2d[0], F2d[1], SampleSpacingZ, X1z[0], X1z[1], &a, &b);
		PSF_Row_Samples[N_int*PixelNumber + idX] = evallinear(a, b, theta[2]);
		dPSFz_Row_Samples[N_int*PixelNumber + idX] = a;

		gencoeff(dF2d_x[0], dF2d_x[1], SampleSpacingZ, X1z[0], X1z[1], &a, &b);
		dPSFx_Row_Samples[N_int*PixelNumber + idX] = evallinear(a, b, theta[2]);

		gencoeff(dF2d_y[0], dF2d_y[1], SampleSpacingZ, X1z[0], X1z[1], &a, &b);
		dPSFy_Row_Samples[N_int*PixelNumber + idX] = evallinear(a, b, theta[2]);

		
		
		__syncthreads();
		//now sum over the row
		if (threadIdx.x == 0) 
		for (int jj = 0; jj < N_int; jj++)
		{
			PSF_Row_Sum[threadIdx.y] += PSF_Row_Samples[N_int*PixelNumber + jj];
			dPSFx_Row_Sum[threadIdx.y] += dPSFx_Row_Samples[N_int*PixelNumber + jj];
			dPSFy_Row_Sum[threadIdx.y] += dPSFy_Row_Samples[N_int*PixelNumber + jj];
			dPSFz_Row_Sum[threadIdx.y] += dPSFz_Row_Samples[N_int*PixelNumber + jj];
		}
	}

	//now return value for each pixel

	__syncthreads();
	if (threadIdx.x == 0)
	{
		PSFs[PSFSizeOut*PSFSizeOut*blockIdx.y + PSFSizeOut*blockIdx.x + threadIdx.y] = PSF_Row_Sum[threadIdx.y];
		dPSFx[PSFSizeOut*PSFSizeOut*blockIdx.y + PSFSizeOut*blockIdx.x + threadIdx.y] = dPSFx_Row_Sum[threadIdx.y];
		dPSFy[PSFSizeOut*PSFSizeOut*blockIdx.y + PSFSizeOut*blockIdx.x + threadIdx.y] = dPSFy_Row_Sum[threadIdx.y];
		dPSFz[PSFSizeOut*PSFSizeOut*blockIdx.y + PSFSizeOut*blockIdx.x + threadIdx.y] = dPSFz_Row_Sum[threadIdx.y];
	}
	

}

__global__ void kernel_calcPSFI(float *psf, float *psfI, float *I, float *bg, int Nfit, int PSFsize)
{
	const int tx = threadIdx.x;
	const int bx = blockIdx.x;
	const int BlockSize = blockDim.x;

	//Prevent read/write past end of array
	int j = BlockSize*bx + tx;
	if ((bx*BlockSize + tx) >= Nfit) return;

	for (int i = 0; i < PSFsize; i++)
	{
		psfI[j*PSFsize + i] = psf[j*PSFsize + i] * I[j] + bg[j];

	}
}

__device__ void gencoeff(float f1, float f2, float dx, float x1, float x2,float *a, float *b)
{
	a[0] = (f2 - f1) / dx;
	b[0] = (x2*f1 - x1*f2) / dx;
}

__device__ float evallinear(float a, float b, float x)
{
	float f;
	f = x*a + b;
	return f;
}