#include "hip/hip_runtime.h"


// includes, system

#include <stdio.h>
// includes, project

//basic includes, others may be needed depending on application
#include <stdlib.h>
#include <string.h>
#include "mex.h"
#include "matrix.h"
#include "hip/hip_runtime.h"

// Thread block size
#define BSZ 128
#define MEM 70
#define IMSZ 11
#define IMSZBIG 21
#define imMEM 4000
#define NK 256 //number of blocks to run in each kernel
#define pi 3.141592
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#define max(a,b)            (((a) > (b)) ? (a) : (b))

//kernel_MLEFit<<<dimGrid, dimBlock>>>(ii, sz, BlockSize, fitnum, d_xarray, d_yarray, d_Narray, d_barray, d_fishermatrix, BlockSize);


__global__ void kernel_guassiansampleblobs(int,int,int, float*,float*,float*, float*,float*,float*,float*,float*,float*);
__global__ void kernel_guassianintegrateblobs(int,int,int, float*,float*,float*, float*,float*,float*,float*,float*,float*);



//__device__ float PSF_xy(float x, int ii, float PSFSigma) {
//    norm=1.0/2.0/PSFSigma/PSFSigma;
//    return 1.0/2.0*(erf((ii-x+0.5)*sqrt(norm))-erf((ii-x-0.5)*sqrt(norm)));
//};
//
//__device__ float MODEL(float *x, float *y, float * Narray, int ii, int jj, float PSFSigma, float b, int N) {
//    float model;
//    model=b;
//    for (nn=0;nn<N;nn++)
//        model+=Narray[nn]*PSF_xy(xarray[nn], ii, PSFSigma)*PSF_xy(yarray[nn], jj, PSFSigma);
//    return model;
//
//};

void CUDAERRROR(const char *instr) {
	hipError_t errornum;
	const char *str;
	if (errornum = hipGetLastError()) {
		str = hipGetErrorString(errornum);
		hipDeviceReset(); //release context so future hipSetDevice calls work
		mexErrMsgIdAndTxt("CudaTemplate:CUDA", "%s: %s\nYou should clear this function in MATLAB for proper operation.\n", instr, str);
	}
}

void mexFunction(int nlhs, mxArray *plhs[],	int	nrhs, const	mxArray	*prhs[]) {
	int blockx;
	int threadx;
	int ii,iii,jj,kk,flag;
	int memblobsnum,ysz,xsz;
	float * xarray, * yarray, * Narray, *bg,*yt,*xl,*xsigma,*ysigma,*covariance,*im;
	float *d_xarray, *d_yarray, *d_Narray, *d_xsigma, *d_ysigma,*d_covariance,*d_im,*d_xl,*d_yt,*subim;
	const mwSize *datasize;
	int locr;
	mwSize imdim[2];



	if (nrhs<9)
		mexErrMsgTxt("xsize,ysize,x_array, y_array, N_array, sigmaX, sigmaY, covariance, UseIntegrated_FLAG\n");

	if (mxGetClassID(prhs[0])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");
	if (mxGetClassID(prhs[1])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");
	if (mxGetClassID(prhs[2])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");
	if (mxGetClassID(prhs[3])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");
	if (mxGetClassID(prhs[4])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");
	if (mxGetClassID(prhs[5])!=mxSINGLE_CLASS)
		mexErrMsgTxt("Data must be comprised of single floats!\n");


	datasize=mxGetDimensions(prhs[2]);
	if (datasize[1]!=1)
		mexErrMsgTxt("xarray should be n X 1 array\n");

	datasize=mxGetDimensions(prhs[3]);

	if (datasize[1]!=1)
		mexErrMsgTxt("xarray should be n X 1 array\n");

	datasize=mxGetDimensions(prhs[4]);

	if (datasize[1]!=1)
		mexErrMsgTxt("xarray should be n X 1 array\n");

	datasize=mxGetDimensions(prhs[5]);
	if (datasize[1]!=1)
		mexErrMsgTxt("xarray should be n X 1 array\n");


	xsz =(float) mxGetScalar(prhs[0]);
	ysz =(float) mxGetScalar(prhs[1]);
	imdim[0]=xsz;
	imdim[1]=ysz;
	//PSFSigma=(float)mxGetScalar(prhs[1]); //matlab-dip_image convention
	xarray =(float *) mxGetData(prhs[2]);
	yarray =(float *) mxGetData(prhs[3]);
	Narray =(float *) mxGetData(prhs[4]);
	xsigma =(float *)mxGetData(prhs[5]);
	ysigma =(float *)mxGetData(prhs[6]);
	covariance =(float *)mxGetData(prhs[7]);
	flag =(float) mxGetScalar(prhs[8]);


	int blobn=datasize[0];
	float maxsigma=-1;
	float sigma;
	for(ii=0;ii<blobn;ii++){
		sigma=sqrt(pow(xsigma[ii],2)+pow(ysigma[ii],2));
		maxsigma=max(maxsigma,sigma);
	}

	int sz=(int) round(float(8*maxsigma));
	sz=min(sz,20);


	if ((flag!=1)&&(flag!=0))
		mexErrMsgTxt("flag can only be 0 or 1\n");

	// over allocate for additional thread reading error
	int BlockSize=min(ceil((float) 15000/4/sz/sz),64);
	memblobsnum=(int)ceil((float)datasize[0]/BlockSize)+128;

	//mexPrintf("Starting CUDA Malloc\n");
	
	CUDAERRROR("P1");

	hipMalloc(&d_xarray, memblobsnum*BlockSize*sizeof(float));
	CUDAERRROR("M1");

	hipMemset(d_xarray, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_xarray, xarray, datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("S1");

	hipMalloc((void**)&d_yarray, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_yarray, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_yarray, yarray,datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("M2");

	hipMalloc((void**)&d_Narray, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_Narray, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_Narray, Narray,datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("M3");

	hipMalloc((void**)&d_xsigma, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_xsigma, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_xsigma, xsigma,datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("M4");

	hipMalloc((void**)&d_ysigma, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_ysigma, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_ysigma, ysigma,datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("M5");

	hipMalloc((void**)&d_covariance, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_covariance, 0, memblobsnum*BlockSize*sizeof(float));
	hipMemcpy(d_covariance, covariance,datasize[0]*sizeof(float), hipMemcpyHostToDevice);
	CUDAERRROR("M6");


	hipMalloc((void**)&d_im, sz*sz*memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_im, 0, sz*sz*memblobsnum*BlockSize*sizeof(float));

	hipMalloc((void**)&d_xl, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_xl, 0, memblobsnum*BlockSize*sizeof(float));

	hipMalloc((void**)&d_yt, memblobsnum*BlockSize*sizeof(float));
	hipMemset(d_yt, 0, memblobsnum*BlockSize*sizeof(float));





	//only run NK blocks in each kernel
	int numK=(int)ceil((float)datasize[0]/BlockSize/NK);

	for (int ii=0;ii<numK;ii++) {

		blockx = min(ceil(((float)(((float)datasize[0])/BlockSize)-ii*NK)), NK);
		blockx = max(blockx,1);
		threadx= BlockSize;


		dim3 dimBlock(threadx);
		dim3 dimGrid(blockx);

		//printf("threadx: %d,blockx: %d\n", threadx, blockx);

		switch (flag)
		{
		case 0:
			kernel_guassiansampleblobs<<<dimGrid, dimBlock>>>(ii,BlockSize,sz, d_xarray,d_yarray,d_Narray, d_xsigma,d_ysigma,d_covariance,d_im,d_xl,d_yt);
			break;//15x15 images, 64 per block
		case 1:
			kernel_guassianintegrateblobs<<<dimGrid, dimBlock>>>(ii,BlockSize,sz, d_xarray,d_yarray,d_Narray, d_xsigma,d_ysigma,d_covariance,d_im,d_xl,d_yt);
			break;//15x15 images, 64 per block
		}

		CUDAERRROR("kernel");
		//mexEvalString("pause(0.001)");

	}

	subim= (float * )malloc(datasize[0]*sz*sz*sizeof(float));
	xl=(float * )malloc(datasize[0]*sizeof(float));
	yt=(float * )malloc(datasize[0]*sizeof(float));


	//reconstruct images
	plhs[0]=mxCreateNumericArray(2, imdim, mxSINGLE_CLASS, mxREAL);
	im=(float *)mxGetData(plhs[0]);

	hipMemcpy(subim, d_im, datasize[0]*sz*sz*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(xl, d_xl, datasize[0]*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(yt, d_yt, datasize[0]*sizeof(float), hipMemcpyDeviceToHost);


	for(kk=0;kk<blobn;kk++){
		for(jj=0;jj<sz;jj++){
			for(iii=0;iii<sz;iii++){
				if ((((int)xl[kk]+iii)<(xsz-1))&&(((int)yt[kk]+jj)<(ysz-1))){
					locr=((int)yt[kk]+jj)*xsz+(int)xl[kk]+iii;
					if((subim[kk*sz*sz+jj*sz+iii]>0)&&(subim[kk*sz*sz+jj*sz+iii]<100000)&&(locr>=0)&&(locr<=((xsz-1)*(ysz))))
					im[locr]+=subim[kk*sz*sz+jj*sz+iii];	
				}
			}
		}
	}



	free(subim);
	free(xl);
	free(yt);
	hipFree(d_xarray);
	hipFree(d_yarray);
	hipFree(d_Narray);
	hipFree(d_xsigma);
	hipFree(d_ysigma);
	hipFree(d_covariance);
	hipFree(d_im);
	hipFree(d_xl);
	hipFree(d_yt);
	hipDeviceReset();

}


//kernel_guassiansampleblobs<<<dimGrid, dimBlock>>>(ii,blockx,BlockSize,sz, d_xarray,d_yarray,d_Narray, d_xsigma,d_ysigma,d_covariance,d_im,d_xl,d_yt);   //15x15 images, 64 per block

__global__ void kernel_guassiansampleblobs(int iiK,int BlockSize, int sz, float *d_xarray,float *d_yarray,float *d_Narray, float *d_xsigma,float *d_ysigma,float *d_covariance,float *d_im,float *d_xl,float *d_yt  ) {
	int tx = threadIdx.x; //matrix number index
	int bx = blockIdx.x;
	float x,y,xsigma,ysigma,covariance,N;
	float xl;
	float yt;
	int ii,jj,pixelx,pixely;


	float model;//

	__shared__ float s_im[imMEM];


	bx=bx+iiK*NK;
	//import datas from device to shared memory

	x=d_xarray[bx*BlockSize+tx];
	y=d_yarray[bx*BlockSize+tx];
	N=d_Narray[bx*BlockSize+tx];
	xsigma=d_xsigma[bx*BlockSize+tx];
	ysigma=d_ysigma[bx*BlockSize+tx];
	covariance=d_covariance[bx*BlockSize+tx];
	xl=round(x)-round(float (sz/2-1));
	xl=max(xl,0);

	yt=round(y)-round(float (sz/2-1));
	yt=max(yt,0);


	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {

		// generate model for pixel ii jj
		pixelx=ii;
		pixely=jj;
		s_im[tx*sz*sz+jj*sz+ii]=N/(2*pi*xsigma*ysigma*sqrt(1-pow(covariance,2)))*exp(-1/(2*(1-pow(covariance,2)))*(pow(x-xl-pixelx,2)/pow(xsigma,2)+pow(y-yt-pixely,2)/pow(ysigma,2)-2*covariance*(x-xl-pixelx)*(y-yt-pixely)/(xsigma*ysigma)));
	}



	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++)
	{
		d_im[bx*BlockSize*sz*sz+tx*sz*sz+jj*sz+ii]=s_im[tx*sz*sz+jj*sz+ii];
		d_xl[bx*BlockSize+tx]=xl;
		d_yt[bx*BlockSize+tx]=yt;
	}

	return;



}



__global__ void kernel_guassianintegrateblobs(int iiK,int BlockSize, int sz, float *d_xarray,float *d_yarray,float *d_Narray, float *d_xsigma,float *d_ysigma,float *d_covariance,float *d_im,float *d_xl,float *d_yt  ) {
	int tx = threadIdx.x; //matrix number index
	int bx = blockIdx.x;
	float x,y,xsigma,ysigma,covariance,N;
	float xl;
	float yt;
	int ii,jj,pixelx,pixely;


	float model;//

	__shared__ float s_im[imMEM];


	bx=bx+iiK*NK;
	//import datas from device to shared memory

	x=d_xarray[bx*BlockSize+tx];
	y=d_yarray[bx*BlockSize+tx];
	N=d_Narray[bx*BlockSize+tx];
	xsigma=d_xsigma[bx*BlockSize+tx];
	ysigma=d_ysigma[bx*BlockSize+tx];
	covariance=d_covariance[bx*BlockSize+tx];

	xl=round(x)-round(float (sz/2-1));
	xl=max(xl,0);

	yt=round(y)-round(float (sz/2-1));
	yt=max(yt,0);

	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {

		// generate model for pixel ii jj
		pixelx=ii;
		pixely=jj;
		s_im[tx*sz*sz+jj*sz+ii]=N/4*(erf((x-xl-pixelx-0.5)/sqrt(2*pow(xsigma,2)))-erf((x-xl-pixelx+0.5)/sqrt(2*pow(xsigma,2))))*(erf((y-yt-pixely-0.5)/sqrt(2*pow(ysigma,2)))-erf((y-yt-pixely+0.5)/sqrt(2*pow(ysigma,2))));  //exp(-1/(2*(1-pow(covariance,2)))*(pow(x-xl-pixelx,2)/pow(xsigma,2)+pow(y-yt-pixely,2)/pow(ysigma,2)-2*covariance*(x-xl-pixelx)*(y-yt-pixely)/(xsigma*ysigma)));
	}



	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++)
	{
		d_im[bx*BlockSize*sz*sz+tx*sz*sz+jj*sz+ii]=s_im[tx*sz*sz+jj*sz+ii];
		d_xl[bx*BlockSize+tx]=xl;
		d_yt[bx*BlockSize+tx]=yt;
	}

	return;



}

//END OF KERNAL FUNCTION


